#include "hip/hip_runtime.h"
/*

The MIT License (MIT)

Copyright (c) 2017 Tim Warburton

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define USE_GRAPH 1


static const int p_Nq = comp_Nq;
static const int p_cubNq = comp_cubNq;

static const int p_halfNq = ((comp_Nq+1)/2);
static const int p_halfCubNq = ((comp_cubNq+1)/2);

static const int p_padCubNq = (p_cubNq%4) ? 0:1;

#define p_Nq2 (p_Nq*p_Nq)
#define p_Np  (p_Nq*p_Nq*p_Nq)

#define p_cubNq2 (p_cubNq*p_cubNq)
#define p_cubNp  (p_cubNq*p_cubNq*p_cubNq)

#define p_Nvgeo 1
#define p_JWID 0

#define p_Nwarps ((p_Nq2+32-1)/32)

#if comp_Nq<=2
#define p_Nblock 8
#warning "using 8 elements per block"
#elif comp_Nq<=4
#define p_Nblock 2
#warning "using 2 elements per block"
#elif comp_Nq<10
#define p_Nblock 1
#warning "using 1 elements per block"
#else
#define p_Nblock 1
#warning "using 1 elements per block"
#endif

#define dlong int
#define hlong dlong
#define dfloat double

__constant__ dfloat const_oddI[p_halfCubNq][p_halfNq];
__constant__ dfloat const_evenI[p_halfCubNq][p_halfNq];


void dfloatRandAlloc(int N, dfloat **h_a, dfloat **c_a){

  *h_a = (dfloat*) calloc(N, sizeof(dfloat));

  for(int n=0;n<N;++n)
    h_a[0][n] = drand48();

  hipMalloc(c_a, N*sizeof(dfloat));

  hipMemcpy(c_a[0], h_a[0], N*sizeof(dfloat), hipMemcpyHostToDevice);

}

__global__ void nothingKernel(){  }

__forceinline__ __device__
void massMatrixMultiplyDevice(const dlong Nelements,
			      const dlong element,
			      const dlong elementId,
			      const dfloat * __restrict__ cubvgeo,
			      const dfloat r_oddI[p_halfCubNq][p_halfNq],
			      const dfloat r_evenI[p_halfCubNq][p_halfNq],
			      dfloat s_Ap[p_Nblock][p_cubNq][p_cubNq][p_cubNq+p_padCubNq],
			      dfloat * __restrict__ r_Ap){
  
  dfloat r_tmpOdd[p_halfCubNq];
  dfloat r_tmpEven[p_halfCubNq];
  
  const int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  // assumes barrier before s_Ap was used last
  
  // transform in 'c'
  {
    const int a = t%p_Nq;
    const int b = t/p_Nq;
    
#pragma unroll p_halfNq
    for(int c=0;c<p_halfNq;++c){
      r_tmpOdd[c]  = r_Ap[c] + r_Ap[p_Nq-1-c];
      r_tmpEven[c] = r_Ap[c] - r_Ap[p_Nq-1-c];
    }
    
#pragma unroll p_halfCubNq
    for(int k=0;k<p_halfCubNq;++k){
      dfloat resOdd = 0, resEven = 0;
      
#pragma unroll p_halfNq
      for(int c=0;c<p_halfNq;++c){
	
	resOdd += r_oddI[k][c]*r_tmpOdd[c];
	resEven += r_evenI[k][c]*r_tmpEven[c];
	
      }
      s_Ap[blk][k][b][a]           = resOdd + resEven;
      s_Ap[blk][p_cubNq-1-k][b][a] = resOdd - resEven;
    }
    
  }
  
  __syncthreads();

  // transform in 'b'
  {
    for(int n=t;n<p_Nq*p_cubNq;n+=p_Nq2){
      const int a = n%p_Nq;
      const int k = n/p_Nq;

#pragma unroll p_halfNq
      for(int b=0;b<p_halfNq;++b){
	dfloat ApOdd  = s_Ap[blk][k][b][a];
	dfloat ApEven = s_Ap[blk][k][p_Nq-1-b][a];
	r_tmpOdd[b]  = ApOdd + ApEven;
	r_tmpEven[b] = ApOdd - ApEven;
      }      
      
#pragma unroll p_halfCubNq
      for(int j=0;j<p_halfCubNq;++j){
	dfloat resOdd = 0, resEven = 0;
	
#pragma unroll p_halfNq
	for(int b=0;b<p_halfNq;++b){
	  resOdd += r_oddI[j][b]*r_tmpOdd[b];
	  resEven += r_evenI[j][b]*r_tmpEven[b];
	}
	
	s_Ap[blk][k][j][a]           = resOdd+resEven;
	s_Ap[blk][k][p_cubNq-1-j][a] = resOdd-resEven;
	
      }
    }

  }
  
  __syncthreads();

  // transform in 'a'
  {
    for(int n=t;n<p_cubNq2;n+=p_Nq2){
      const int j = n%p_cubNq;
      const int k = n/p_cubNq;
      
#pragma unroll p_halfNq
      for(int a=0;a<p_halfNq;++a){
	dfloat ApOdd  = s_Ap[blk][k][j][a];
	dfloat ApEven = s_Ap[blk][k][j][p_Nq-1-a];
	r_tmpOdd[a]  = ApOdd + ApEven;
	r_tmpEven[a] = ApOdd - ApEven;
      }
      
#pragma unroll p_halfCubNq
      for(int i=0;i<p_halfCubNq;++i){
	dfloat resOdd = 0, resEven = 0;
	
#pragma unroll p_halfNq
	for(int a=0;a<p_halfNq;++a){
	  resOdd  += r_oddI[i][a]*r_tmpOdd[a];
	  resEven += r_evenI[i][a]*r_tmpEven[a];
	}

	dlong gid1 = elementId*p_cubNp + k*p_cubNq*p_cubNq + j*p_cubNq + i;
	dlong gid2 = elementId*p_cubNp + k*p_cubNq*p_cubNq + j*p_cubNq + p_cubNq-1-i;
	
	dfloat WJ1 = (element<Nelements) ? cubvgeo[gid1]: 0;
	dfloat WJ2 = (element<Nelements) ? cubvgeo[gid2]: 0;
	
	dfloat ApOdd = WJ1*(resOdd + resEven);
	dfloat ApEven = WJ2*(resOdd - resEven);

	r_Ap[i] = ApOdd + ApEven;
	r_Ap[p_cubNq-1-i] = ApOdd - ApEven;
      }

#pragma unroll p_halfNq
      for(int a=0;a<p_halfNq;++a){
	dfloat resOdd = 0, resEven = 0;
	
#pragma unroll p_halfCubNq
	for(int i=0;i<p_halfCubNq;++i){
	  resOdd  += r_oddI[i][a]*r_Ap[i];
	  resEven += r_evenI[i][a]*r_Ap[p_cubNq-1-i];
	}
	
	s_Ap[blk][k][j][a]        = resOdd + resEven;
	s_Ap[blk][k][j][p_Nq-1-a] = resOdd - resEven;
      }
    }
  }
  
  __syncthreads();

  
  // test in 'b'
  {

    for(int n=t;n<p_Nq*p_cubNq;n+=p_Nq2){
      const int a = n%p_Nq;
      const int k = n/p_Nq;

      for(int j=0;j<p_halfCubNq;++j){
	dfloat ApOdd  = s_Ap[blk][k][j][a];
	dfloat ApEven = s_Ap[blk][k][p_cubNq-1-j][a];
	r_tmpOdd[j]  = ApOdd + ApEven;
	r_tmpEven[j] = ApOdd - ApEven;
      }

#pragma unroll p_halfNq
      for(int b=0;b<p_halfNq;++b){
	dfloat resOdd = 0, resEven = 0;
	
#pragma unroll p_halfCubNq
	for(int j=0;j<p_halfCubNq;++j){
	  resOdd  += r_oddI[j][b]*r_tmpOdd[j];
	  resEven += r_evenI[j][b]*r_tmpEven[j];
	}
	
	s_Ap[blk][k][b][a]        = resOdd + resEven;
	s_Ap[blk][k][p_Nq-1-b][a] = resOdd - resEven;
      }
    }
  }
  
  __syncthreads();

  // test in 'c'
  {
    const int a = t%p_Nq;
    const int b = t/p_Nq;

    for(int k=0;k<p_halfCubNq;++k){
      dfloat ApOdd  = s_Ap[blk][k][b][a];
      dfloat ApEven = s_Ap[blk][p_cubNq-1-k][b][a];
      r_tmpOdd[k]  = ApOdd + ApEven;
      r_tmpEven[k] = ApOdd - ApEven;
    }
    
#pragma unroll p_halfNq
    for(int c=0;c<p_halfNq;++c){
      dfloat resOdd = 0, resEven = 0;
      
#pragma unroll p_halfCubNq
      for(int k=0;k<p_halfCubNq;++k){
	resOdd  += r_oddI[k][c]*r_tmpOdd[k];
	resEven += r_evenI[k][c]*r_tmpEven[k];
      }
      
      r_Ap[c]        = resOdd + resEven;
      r_Ap[p_Nq-1-c] = resOdd - resEven;
      
    }
  }

}


__global__ void massMatrixMultiplyRegisterKernel(const dlong Nelements,
						 const dlong  * __restrict__ elementIds,
						 const dfloat * __restrict__ cubvgeo,
						 const dfloat * __restrict__ oddI,
						 const dfloat * __restrict__ evenI,
						 const dfloat * __restrict__ q,
						 dfloat * __restrict__ qnew){
  
  __shared__ dfloat s_tmp1[p_Nblock][p_cubNq][p_cubNq][p_cubNq+p_padCubNq];
  __shared__ dfloat s_oddI[p_halfNq*p_halfCubNq];
  __shared__ dfloat s_evenI[p_halfCubNq*p_halfNq];

  dfloat r_oddI[p_halfCubNq][p_halfNq];
  dfloat r_evenI[p_halfCubNq][p_halfNq];

  dfloat r_Aq[p_cubNq];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const dlong e = blockIdx.x*p_Nblock + blk;

  const dlong element = (e<Nelements) ? elementIds[e]: 0;
  
  const unsigned int a = t%p_Nq;
  const unsigned int b = t/p_Nq;

  for(int c=0;c<p_Nq;++c){
    
    dlong id = a + b*p_Nq + c*p_Nq2 + element*p_Np;
    
    r_Aq[c] = q[id];
  }

  for(int n=t;n<p_halfNq*p_halfCubNq;n+=p_Nq*p_Nq){
    s_oddI[n] = oddI[n];
    s_evenI[n] = evenI[n];
    n+=p_Nq*p_Nq;
  }

  __syncthreads();
  
  for(int n=0;n<p_halfNq*p_halfCubNq;++n){
    r_oddI[0][n] = s_oddI[n];
    r_evenI[0][n] = s_evenI[n];
  }
  
  massMatrixMultiplyDevice(Nelements, e, element, cubvgeo, r_oddI, r_evenI, s_tmp1, r_Aq);
  
  if(e<Nelements){
#pragma unroll p_Nq
    for(int c=0;c<p_Nq;++c){
      dlong id = a + b*p_Nq + c*p_Nq2 + element*p_Np;
      qnew[id] = r_Aq[c];
    }
  }
}



__global__ void massMatrixMultiplySharedKernel(const dlong Nelements,
						 const dlong  * __restrict__ elementIds,
						 const dfloat * __restrict__ cubvgeo,
						 const dfloat * __restrict__ oddI,
						 const dfloat * __restrict__ evenI,
						 const dfloat * __restrict__ q,
						 dfloat * __restrict__ qnew){
  
  __shared__ dfloat s_tmp1[p_Nblock][p_cubNq][p_cubNq][p_cubNq+p_padCubNq];
  __shared__ dfloat s_oddI[p_halfCubNq][p_halfNq];
  __shared__ dfloat s_evenI[p_halfCubNq][p_halfNq];

  dfloat r_Aq[p_cubNq];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const dlong e = blockIdx.x*p_Nblock + blk;

  const dlong element = (e<Nelements) ? elementIds[e]: 0;
  
  const unsigned int a = t%p_Nq;
  const unsigned int b = t/p_Nq;

  for(int c=0;c<p_Nq;++c){
    
    dlong id = a + b*p_Nq + c*p_Nq2 + element*p_Np;
    
    r_Aq[c] = q[id];
  }

  for(int n=t;n<p_halfNq*p_halfCubNq;n+=p_Nq*p_Nq){
    s_oddI[0][n] = oddI[n];
    s_evenI[0][n] = evenI[n];
    n+=p_Nq*p_Nq;
  }

  __syncthreads();
  
  massMatrixMultiplyDevice(Nelements, e, element, cubvgeo, s_oddI, s_evenI, s_tmp1, r_Aq);
  
  if(e<Nelements){
#pragma unroll p_Nq
    for(int c=0;c<p_Nq;++c){
      dlong id = a + b*p_Nq + c*p_Nq2 + element*p_Np;
      qnew[id] = r_Aq[c];
    }
  }
}


__global__ void massMatrixMultiplyConstantKernel(const dlong Nelements,
						 const dlong  * __restrict__ elementIds,
						 const dfloat * __restrict__ cubvgeo,
						 const dfloat * __restrict__ oddI,
						 const dfloat * __restrict__ evenI,
						 const dfloat * __restrict__ q,
						 dfloat * __restrict__ qnew){
  
  __shared__ dfloat s_tmp1[p_Nblock][p_cubNq][p_cubNq][p_cubNq+p_padCubNq];

  dfloat r_Aq[p_cubNq];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const dlong e = blockIdx.x*p_Nblock + blk;

  const dlong element = (e<Nelements) ? elementIds[e]: 0;
  
  const unsigned int a = t%p_Nq;
  const unsigned int b = t/p_Nq;

  for(int c=0;c<p_Nq;++c){
    
    dlong id = a + b*p_Nq + c*p_Nq2 + element*p_Np;
    
    r_Aq[c] = q[id];
  }

  __syncthreads();
  
  massMatrixMultiplyDevice(Nelements, e, element, cubvgeo, const_oddI, const_evenI, s_tmp1, r_Aq);
  
  if(e<Nelements){
#pragma unroll p_Nq
    for(int c=0;c<p_Nq;++c){
      dlong id = a + b*p_Nq + c*p_Nq2 + element*p_Np;
      qnew[id] = r_Aq[c];
    }
  }
}








void massMatrixMultiplyHost(const dlong Nelements,
			    const dlong  * __restrict__ elementIds,
			    const dfloat * __restrict__ cubvgeo,
			    const dfloat * __restrict__ cubI,
			    const dfloat * __restrict__ q,
			    dfloat * __restrict__ qnew){


  dfloat qXXX[p_Nq][p_Nq][p_Nq];
  dfloat qIXX[p_cubNq][p_Nq][p_Nq];
  dfloat qIIX[p_cubNq][p_cubNq][p_Nq];
  dfloat qIII[p_cubNq][p_cubNq][p_cubNq];
    
  for(dlong e=0;e<Nelements;++e){

    for(int c=0;c<p_Nq;++c){
      for(int b=0;b<p_Nq;++b){
	for(int a=0;a<p_Nq;++a){
	  int id = e*p_Np + c*p_Nq2 + b*p_Nq + a;
	  qXXX[c][b][a] = q[id];
	}
      }
    }
    
    for(int k=0;k<p_cubNq;++k){
      for(int b=0;b<p_Nq;++b){
	for(int a=0;a<p_Nq;++a){
	  
	  dfloat res = 0;
	  
	  for(int c=0;c<p_Nq;++c){
	    dfloat Ikc = cubI[k*p_Nq+c];
	    res += Ikc*qXXX[c][b][a];
	  }
	  
	  qIXX[k][b][a] = res;
	}
      }
    }
    
    // interpolate in b
    for(int k=0;k<p_cubNq;++k){
      for(int j=0;j<p_cubNq;++j){
	for(int a=0;a<p_Nq;++a){
	  
	  dfloat res = 0;
	  
	  for(int b=0;b<p_Nq;++b){
	    dfloat Ijb = cubI[j*p_Nq+b];
	    res += Ijb*qIXX[k][b][a];
	  }
	  
	  qIIX[k][j][a] = res;
	}
      }
    }

    // interpolate in a
    for(int k=0;k<p_cubNq;++k){
      for(int j=0;j<p_cubNq;++j){
	for(int i=0;i<p_cubNq;++i){

	  dfloat res = 0;
	  
	  for(int a=0;a<p_Nq;++a){
	    dfloat Iia = cubI[i*p_Nq+a];
	    res += Iia*qIIX[k][j][a];
	  }
	  
	  int gid = e*p_cubNp + k*p_cubNq2 + j*p_cubNq + i;
	  
	  dfloat JW = cubvgeo[gid];

	  qIII[k][j][i] = res*JW;
	}
      }
    }


    // project in a
    for(int k=0;k<p_cubNq;++k){
      for(int j=0;j<p_cubNq;++j){
	for(int a=0;a<p_Nq;++a){

	  dfloat res = 0;
	  
	  for(int i=0;i<p_cubNq;++i){
	    dfloat Iia = cubI[i*p_Nq+a];
	    res += Iia*qIII[k][j][i];
	  }

	  qIIX[k][j][a] = res;
	}
      }
    }


    // project in b
    for(int k=0;k<p_cubNq;++k){
      for(int b=0;b<p_Nq;++b){
	for(int a=0;a<p_Nq;++a){

	  dfloat res = 0;

	  for(int j=0;j<p_cubNq;++j){
	    dfloat Ijb = cubI[j*p_Nq+b];
	    res += Ijb*qIIX[k][j][a];
	  }
	  
	  qIXX[k][b][a] = res;

	}
      }
    }


    // project in c
    for(int c=0;c<p_Nq;++c){
      for(int b=0;b<p_Nq;++b){
	for(int a=0;a<p_Nq;++a){

	  dfloat res = 0;

	  for(int k=0;k<p_cubNq;++k){
	    dfloat Ikc = cubI[k*p_Nq+c];
	    res += Ikc*qIXX[k][b][a];
	  }

	  int id = e*p_Np + c*p_Nq2 + b*p_Nq + a;
	  qnew[id] = res;
	}
      }
    }
  }
    
  
}


void buildInterpMatrices(dfloat *h_I,  dfloat **c_oddI, dfloat **c_evenI){

#if 0
  // now overwrite h_I and copy to c_I
  printf("I = [\n");
  for(int i=0;i<p_cubNq;++i){
    for(int a=0;a<p_Nq;++a){
      printf("% .4e ", h_I[i*p_Nq+a]);
    }
    printf("\n");
  }
  printf("];\n");
#endif
  
  //  hipMemcpy(*c_I, *h_I, p_Nq*p_cubNq*sizeof(dfloat), hipMemcpyHostToDevice);

  dfloat *X = (dfloat*) calloc(p_Nq*p_Nq, sizeof(dfloat));
  dfloat *invX = (dfloat*) calloc(p_Nq*p_Nq, sizeof(dfloat));

  dfloat *cubX = (dfloat*) calloc(p_cubNq*p_cubNq, sizeof(dfloat));
  dfloat *cubInvX = (dfloat*) calloc(p_cubNq*p_cubNq, sizeof(dfloat));

  for(int n=0;n<p_cubNq;++n){
    cubX[n*p_cubNq + n] = 1;
    cubInvX[n*p_cubNq + n] = 0.5;

    if(n<p_cubNq/2){
      cubX[n*p_cubNq + p_cubNq-1-n] = -1;
      cubInvX[n*p_cubNq + p_cubNq-1-n] = +0.5;
    }
    
    if(n>=(p_cubNq/2)){
      cubX[n*p_cubNq + p_cubNq-1-n] = +1;
      cubInvX[n*p_cubNq + p_cubNq-1-n] = -0.5;
    }
  }

  for(int n=0;n<p_Nq;++n){
    X[n*p_Nq + n] = 1;
    invX[n*p_Nq + n] = 0.5;

    if(n<p_Nq/2){
      X[n*p_Nq + p_Nq-1-n] = 1;
      invX[n*p_Nq + p_Nq-1-n] = -0.5;
    }
    
    if(n>=p_Nq/2){
      X[n*p_Nq + p_Nq-1-n] = -1;
      invX[n*p_Nq + p_Nq-1-n] = 0.5;
    }
  }

  if(p_Nq%2) invX[(p_Nq)*(p_Nq)/2] = 1;
  if(p_cubNq%2) cubInvX[(p_cubNq+1)*(p_cubNq+1)/2] = 1;
  
  dfloat *IinvX = (dfloat*) calloc(p_Nq*p_cubNq, sizeof(dfloat));
  dfloat *cubInvXIinvX = (dfloat*) calloc(p_Nq*p_cubNq, sizeof(dfloat));

  // post multiply by invX
  for(int i=0;i<p_cubNq;++i){
    for(int a=0;a<p_Nq;++a){
      dfloat res = 0;
      for(int n=0;n<p_Nq;++n){
	res += h_I[i*p_Nq+n]*invX[n*p_Nq+a];
      }
      IinvX[i*p_Nq+a] = res;
    }
  }

  // pre multiply by invX
  for(int i=0;i<p_cubNq;++i){
    for(int a=0;a<p_Nq;++a){
      dfloat res = 0;
      for(int n=0;n<p_cubNq;++n){
	res += cubInvX[i*p_cubNq+n]*IinvX[n*p_Nq + a];
      }
      cubInvXIinvX[i*p_Nq+a] = res;
    }
  }

  // now interleave the two non-zero blocks
  // [ A 0 ]  => [ A[0][0] B[0][0] A[0][1] B[0][1] .. A[0][p_halfNq-1] B[0][p_halfNq-1] .. 
  // [ 0 B ] 

  dfloat *oddI  = (dfloat*) calloc(p_cubNq*p_halfCubNq, sizeof(dfloat));
  dfloat *evenI = (dfloat*) calloc(p_cubNq*p_halfCubNq, sizeof(dfloat));
  
  for(int i=0;i<p_halfCubNq;++i){
    for(int a=0;a<p_halfNq;++a){

      oddI[i*p_halfNq+a] = cubInvXIinvX[i*p_Nq+a];
      evenI[i*p_halfNq+a] = cubInvXIinvX[(p_cubNq-1-i)*p_Nq + p_Nq-1-a];
      
    }
  }
      
  int NoddI = p_halfCubNq*p_halfNq;
  int NevenI = p_halfCubNq*p_halfNq;
  
  hipMalloc(c_oddI, NoddI*sizeof(dfloat));
  hipMalloc(c_evenI, NevenI*sizeof(dfloat));
  
  hipMemcpy(*c_oddI,  oddI,  NoddI*sizeof(dfloat),  hipMemcpyHostToDevice);
  hipMemcpy(*c_evenI, evenI, NoddI*sizeof(dfloat), hipMemcpyHostToDevice);
  
  hipMemcpyToSymbol(HIP_SYMBOL(const_oddI),  oddI,  NoddI*sizeof(dfloat));
  hipMemcpyToSymbol(HIP_SYMBOL(const_evenI), evenI, NoddI*sizeof(dfloat));
}


int main(int argc, char **argv){

  hipStream_t stream;
  hipStreamCreate(&stream);
  
  if(argc!=2){
    printf("Usage: ./massMatrixMultiplyVT Nelements\n");
    exit(-1);
  }

  // read number of elements
  hlong Nelements = atoi(argv[argc-1]);
  
  int    Ntotal = Nelements*p_Np;
  int cubNtotal = Nelements*p_cubNp;

  dfloat *h_cubvgeo, *c_cubvgeo;
  dfloat *h_qnew,    *c_qnew;
  dfloat *h_q,       *c_q;
  dfloat *h_I,       *c_I;
  dfloat *c_oddI,    *c_evenI;
  dfloat *h_garbage, *c_garbage;
  int    *h_elementIds, *c_elementIds;

  // list of elements
  h_elementIds = (int*) calloc(Nelements, sizeof(int));
  for(int e=0;e<Nelements;++e)
    h_elementIds[e] = e;
  hipMalloc(&c_elementIds, Nelements*sizeof(int));
  hipMemcpy(c_elementIds, h_elementIds, Nelements*sizeof(int), hipMemcpyHostToDevice);
  
  // float fields
  dfloatRandAlloc(cubNtotal*p_Nvgeo, &h_cubvgeo, &c_cubvgeo);

  for(int e=0;e<Nelements;++e){
    for(int n=0;n<p_cubNp;++n){
      h_cubvgeo[e*p_cubNp+n] = drand48();
    }
  }

  hipMemcpy(c_cubvgeo, h_cubvgeo, p_Nvgeo*Nelements*p_cubNp*sizeof(dfloat), hipMemcpyHostToDevice);
  
  dfloatRandAlloc(Ntotal,       &h_q, &c_q);
  dfloatRandAlloc(Ntotal,       &h_qnew, &c_qnew);

  dfloatRandAlloc(p_Nq*p_cubNq, &h_I, &c_I);
  
  // give I the correct symmetry
  for(int i=0;i<p_halfCubNq;++i){
    for(int a=0;a<p_Nq;++a){
      h_I[(p_cubNq-1-i)*p_Nq + p_Nq-1-a] = h_I[i*p_Nq+a];
    }
  }

  // create Odd-even packed storage for I and transpose(I) and push to constant memory
  buildInterpMatrices(h_I, &c_oddI, &c_evenI);

  // flush L2 ??
  int sz = 32*1024*1024; // 32MB
  dfloatRandAlloc(sz, &h_garbage, &c_garbage);
  
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);	

  int Ntests = 100;
  // KERNEL GRID
  dim3 G((Nelements+p_Nblock-1)/p_Nblock, 1, 1);
  dim3 B(p_Nq*p_Nq, p_Nblock, 1);

  float nothingElapsed = 0;
  {

    // time kernel that does nothing

#if USE_GRAPH==1
    // cuda stream capture sequence for nothingKernel
    hipGraph_t nothingGraph;
  
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    
    for(int test=0;test<Ntests;++test){
      nothingKernel <<< 1, 1, 0, stream >>> ();
    }
    
    hipStreamEndCapture(stream, &nothingGraph);

    // time graph sequence for nothing
    hipGraphExec_t nothingInstance;
    hipGraphInstantiate(&nothingInstance, nothingGraph, NULL, NULL, 0);
    
    hipEventRecord(start, stream);
    
    hipGraphLaunch(nothingInstance, stream);
    
    hipEventRecord(end, stream);
#else
    
    hipEventRecord(start, stream);
    
    for(int test=0;test<Ntests;++test)
    nothingKernel <<< 1, 1, 0, stream >>> ();
    
    hipEventRecord(end, stream);
    
#endif
    
    hipDeviceSynchronize();
    
    hipEventElapsedTime(&nothingElapsed, start, end);
    nothingElapsed /= 1000.;
    nothingElapsed /= (double) Ntests;
  
  }
  
  int cacheSwitchNq = 8;
  // warm up call
  if(p_Nq<=cacheSwitchNq) 
    massMatrixMultiplyRegisterKernel <<< G, B, 0, stream >>>(Nelements, c_elementIds, c_cubvgeo, c_oddI, c_evenI, c_q, c_qnew);
  else
    massMatrixMultiplyConstantKernel <<< G, B, 0, stream >>>(Nelements, c_elementIds, c_cubvgeo, c_oddI, c_evenI, c_q, c_qnew);

#if USE_GRAPH==1
  // cuda stream capture
  hipGraph_t graph;
  
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  for(int test=0;test<Ntests;++test){

    if(p_Nq<=cacheSwitchNq) 
      massMatrixMultiplyRegisterKernel <<< G, B, 0, stream >>>(Nelements, c_elementIds, c_cubvgeo, c_oddI, c_evenI, c_q, c_qnew);
    else
      massMatrixMultiplyConstantKernel <<< G, B, 0, stream >>>(Nelements, c_elementIds, c_cubvgeo, c_oddI, c_evenI, c_q, c_qnew);
    
  }

  hipStreamEndCapture(stream, &graph);
  
  hipGraphExec_t instance;
  hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
#endif
  
  hipDeviceSynchronize();

  {
    hipEventRecord(start, stream);
    
#if USE_GRAPH==0
    for(int test=0;test<Ntests;++test){

      if(p_Nq<=cacheSwitchNq) 
	massMatrixMultiplyRegisterKernel <<< G, B, 0, stream >>>(Nelements, c_elementIds, c_cubvgeo, c_oddI, c_evenI, c_q, c_qnew);
      else
	massMatrixMultiplyConstantKernel <<< G, B, 0, stream >>>(Nelements, c_elementIds, c_cubvgeo, c_oddI, c_evenI, c_q, c_qnew);
      
    }
#else
    hipGraphLaunch(instance, stream);
#endif

    hipEventRecord(end, stream);
    
    hipEventSynchronize(end);
    
    float elapsed;
    hipEventElapsedTime(&elapsed, start, end);
    elapsed /= 1000.;
    elapsed /= (double) Ntests;
    
    printf("%d %d %d %lg %lg %lg %%%% [MassMatrixMultiply: N, Nelements, Ndofs, elapsed, dofsPerSecond, nothingElapsed]\n", p_Nq-1, Nelements, p_Np*Nelements, elapsed, Nelements*(p_Np/elapsed), nothingElapsed);
  }

  // check output is correct
  massMatrixMultiplyHost(Nelements, h_elementIds, h_cubvgeo, h_I, h_q, h_qnew);

  // copy device version to host old q
  dfloat *fromDevice = (dfloat*) calloc(Nelements*p_Np, sizeof(dfloat));
  hipMemcpy(fromDevice, c_qnew, Nelements*p_Np*sizeof(dfloat), hipMemcpyDeviceToHost);

  dfloat maxDiff = 0;
  
  for(int e=0;e<Nelements;++e){
    for(int n=0;n<p_Np;++n){
      int id = e*p_Np + n;
      dfloat diff = fabs(h_qnew[id]-fromDevice[id]);
      maxDiff = (diff>maxDiff) ? diff:maxDiff;
    }
  }
  printf("|| Mq_{host} - Mq_{device} ||_linf = %lg\n", maxDiff);
  
  hipEventDestroy(start);
  hipEventDestroy(end);	
  
  return 0;

}
